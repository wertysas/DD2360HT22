

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>

#include <random>

#define DataType double
#define TPB 256

using uint = unsigned int;

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, uint len) {
  const uint idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < len) {
    out[idx] =  in1[idx] + in2[idx];
  }
}

// Timer function
double time() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main(int argc, char **argv) {
  uint inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1; DataType *deviceInput2;
  DataType *deviceOutput;

  // Input length reading
  if (argc != 2) {
    printf("Error wrong format! Correct Usage :\t ./vectoradd [array length]");
    return -1;
  }
  inputLength = atoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  // Host memory allocation
  size_t vsizeB =  inputLength*sizeof(double);
  hostInput1 = (DataType*) malloc(vsizeB);
  hostInput2 = (DataType*) malloc(vsizeB);
  hostOutput = (DataType*) malloc(vsizeB);
  resultRef = (DataType*) malloc(vsizeB);
  
  // Initialisation of host arrays to random values
  std::default_random_engine e{};
  std::uniform_real_distribution<DataType> d{-1.0, 1.0};
  for (uint i=0; i<inputLength; i++) { hostInput1[i] = d(e); }
  for (uint i=0; i<inputLength; i++) { hostInput2[i] = d(e); }
  
  // Computation of reference result on host
  double t0 = time();
  for (uint i=0; i<inputLength; i++) {
    resultRef[i] =  hostInput1[i] + hostInput2[i];
  }
  double cpuTiming = time() - t0;

  // GPU memory allocation
  hipMalloc(&deviceInput1, vsizeB);
  hipMalloc(&deviceInput2, vsizeB);
  hipMalloc(&deviceOutput, vsizeB);

  // Copy memory to the GPU
  double tb = time();
  hipMemcpy(&deviceInput1, hostInput1, vsizeB, hipMemcpyHostToDevice);
  hipMemcpy(&deviceInput2, hostInput2, vsizeB, hipMemcpyHostToDevice);

  // Initialize the 1D grid and block dimensions
  uint blockSize = TPB;
  uint gridSize  = (inputLength+blockSize-1) / blockSize;

  // Launch the GPU Kernel
  t0 = time();
  vecAdd<<<gridSize, blockSize>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();
  double gpuTiming = time() - t0;

  // Copy the GPU memory back to the CPU
  hipMemcpy(&hostOutput, &deviceOutput, vsizeB, hipMemcpyDeviceToHost);
  double totalTiming = time() - tb;

  // Compare the output with the reference
  DataType max_diff = 1e-7;
  for (uint i=0; i<inputLength; i++) {
    if (abs(hostOutput[i]-resultRef[i])<1e-7) {
      printf("Error results differ more than maximum value (>%f)\n", max_diff);
      printf("Host Calculated Value: %f\n", resultRef[i]);
      printf("Device Calculated Value: %f\n", hostOutput[i]);
    }
  }
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);
  printf("CPU vector addition time: %f", cpuTiming);
  printf("GPU vector addition time: %f", gpuTiming);
  printf("GPU vector addition + data transfer time: %f", totalTiming);
  return 0;
}
