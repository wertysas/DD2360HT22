

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>

#include <random>

#define DataType double
#define TPB 256

using uint = unsigned int;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, uint len) {
  const uint idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < len) {
    out[idx] =  in1[idx] + in2[idx];
  }
}

// Timer function
double time() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec*1e3 + (double)tp.tv_usec*1e-3);
}


int main(int argc, char **argv) {
  uint inputLength;
  DataType *resultRef;
  DataType *deviceInput1; DataType *deviceInput2;
  DataType *deviceOutput;

  // Input length reading
  if (!(argc == 2 || argc ==3)) {
    printf("Error wrong format! Correct Usage :\t ./vectoradd [vector length] [mode]\n \
            vector length:\n\tpositive integer\n \
            modes:\n\t0 (default output)\n\t1 (csv formatted output for measurements)\n");
    return -1;
  }
  inputLength = atoi(argv[1]);
  bool formattedPrint = false;
  if (argc==3 && atoi(argv[2])==1) {
    formattedPrint = true;
  }

  // Host memory allocation
  size_t vsizeB =  inputLength*sizeof(DataType);
  resultRef = (DataType*) malloc(vsizeB);
  
   
  // GPU memory allocation
  hipMalloc(&deviceInput1, vsizeB);
  hipMalloc(&deviceInput2, vsizeB);
  hipMalloc(&deviceOutput, vsizeB);

  // Pinned Memory allocation
  DataType *pinnedInput1, *pinnedInput2, *pinnedOutput;
  checkCuda(hipHostMalloc( (void**) &pinnedInput1, vsizeB, hipHostMallocDefault) );
  checkCuda(hipHostMalloc( (void**) &pinnedInput2, vsizeB, hipHostMallocDefault) );
  checkCuda(hipHostMalloc( (void**) &pinnedOutput, vsizeB, hipHostMallocDefault) );
  
  // Initialisation of pinned host arrays to random values
  std::default_random_engine e{};
  std::uniform_real_distribution<DataType> d{-1.0, 1.0};
  for (uint i=0; i<inputLength; i++) { pinnedInput1[i] = d(e); }
  for (uint i=0; i<inputLength; i++) { pinnedInput2[i] = d(e); }
  
  // Computation of reference result on host
  double t0 = time();
  for (uint i=0; i<inputLength; i++) {
    resultRef[i] =  pinnedInput1[i] + pinnedInput2[i];
  }
  double cpuTiming = time() - t0;

  // Create and initialise streams
  const int nStreams = 4;
  hipStream_t streams[nStreams];
  const int streamSize = (vsizeB + nStreams - 1) / nStreams;
  const int lastStreamSize = streamSize*nStreams-vsizeB;
  int offset;
  for (int i=0; i<nStreams; i++) {
    checkCuda( hipStreamCreate(&streams[i]) );
  }
  
  // Initialize the 1D grid and block dimensions
  uint blockSize = TPB;
  uint gridSize  = (inputLength+blockSize-1) / blockSize;

  // Copy and kernel execution
  t0 = time();
  for (int i=0; i<nStreams-1; i++) {
    offset= i*streamSize;
    checkCuda( hipMemcpyAsync(&deviceInput1[offset], &pinnedInput1[offset], streamSize, hipMemcpyHostToDevice, streams[i]) );
    checkCuda( hipMemcpyAsync(&deviceInput2[offset], &pinnedInput2[offset], streamSize, hipMemcpyHostToDevice, streams[i]) );
    vecAdd<<<gridSize, blockSize, 0, streams[i]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);
    checkCuda( hipMemcpyAsync(&pinnedOutput[offset], &deviceOutput[offset], streamSize, hipMemcpyDeviceToHost, streams[i]) );
  }
  int i = nStreams-1;
  offset= i*streamSize;
  checkCuda( hipMemcpyAsync(&deviceInput1[offset], &pinnedInput1[offset], lastStreamSize, hipMemcpyHostToDevice, streams[i]) );
  checkCuda( hipMemcpyAsync(&deviceInput2[offset], &pinnedInput2[offset], lastStreamSize, hipMemcpyHostToDevice, streams[i]) );
  vecAdd<<<gridSize, blockSize, 0, streams[nStreams-1]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], lastStreamSize);
  checkCuda( hipMemcpyAsync(&pinnedOutput[offset], &deviceOutput[offset], lastStreamSize, hipMemcpyDeviceToHost, streams[i]) );

  // Copy memory to the GPU
  double vectorAddTiming = time() - t0;
  

  // Compare the output with the reference
  DataType max_diff = 1e-7;
  for (uint i=0; i<inputLength; i++) {
    if (abs(pinnedOutput[i]-resultRef[i])>1e-7) {
      printf("Error results differ more than maximum value (>%f)\n", max_diff);
      printf("Host Calculated Value: %f\n", resultRef[i]);
      printf("Device Calculated Value: %f\n", pinnedOutput[i]);
    }
  }
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  hipHostFree(pinnedInput1);
  hipHostFree(pinnedInput2);
  hipHostFree(pinnedOutput);


  // Measurement prints
  if (formattedPrint) {
    printf("%d, %11.8f, %11.8f\n", inputLength, cpuTiming, vectorAddTiming);
  }
  else {
    printf("Vector length: %d\n", inputLength);
    printf("CPU vector addition time: %f\n", cpuTiming);
    printf("GPU vector addition time: %f\t(incl. data transfer)\n", vectorAddTiming);
  }
  return 0;
}
