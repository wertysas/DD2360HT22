

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include <random>

#define DataType double
#define TPB 256

using uint = unsigned int;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, uint len) {
  const uint idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < len) {
    out[idx] =  in1[idx] + in2[idx];
  }
}

// Timer function
double time() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec*1e3 + (double)tp.tv_usec*1e-3);
}


int main(int argc, char **argv) {
  uint inputLength;

  // Input length reading
  if (!(argc == 2 || argc ==3)) {
    printf("Error wrong format! Correct Usage :\t ./vectoradd [vector length] [mode]\n \
            vector length:\n\tpositive integer\n \
            modes:\n\t0 (default output)\n\t1 (csv formatted output for measurements)\n");
    return -1;
  }
  inputLength = atoi(argv[1]);
  bool formattedPrint = false;
  if (argc==3 && atoi(argv[2])==1) {
    formattedPrint = true;
  }

  // Host memory allocation
  DataType *resultRef;
  size_t vsizeB =  inputLength*sizeof(DataType);
  resultRef = (DataType*) malloc(vsizeB);
  
   
  // GPU memory allocation
  DataType *deviceInput1, *deviceInput2, *deviceOutput;
  hipMalloc(&deviceInput1, vsizeB);
  hipMalloc(&deviceInput2, vsizeB);
  hipMalloc(&deviceOutput, vsizeB);

  // Pinned Memory allocation
  DataType *pinnedInput1, *pinnedInput2, *pinnedOutput;
  checkCuda(hipHostMalloc( (void**) &pinnedInput1, vsizeB, hipHostMallocDefault) );
  checkCuda(hipHostMalloc( (void**) &pinnedInput2, vsizeB, hipHostMallocDefault) );
  checkCuda(hipHostMalloc( (void**) &pinnedOutput, vsizeB, hipHostMallocDefault) );
  
  // Initialisation of pinned host arrays to random values
  std::default_random_engine e{};
  std::uniform_real_distribution<DataType> d{-1.0, 1.0};
  for (uint i=0; i<inputLength; i++) { pinnedInput1[i] = d(e); }
  for (uint i=0; i<inputLength; i++) { pinnedInput2[i] = d(e); }
  
  // Computation of reference result on host
  double t0 = time();
  for (uint i=0; i<inputLength; i++) {
    resultRef[i] =  pinnedInput1[i] + pinnedInput2[i];
  }
  double cpuTiming = time() - t0;

  // Create and initialise streams
  const int nStreams = 4;
  hipStream_t streams[nStreams];
  const int streamSize = (inputLength + nStreams - 1) / nStreams;
  const int lastStreamSize = streamSize*nStreams-inputLength;
  const int streamBytes = sizeof(DataType) * streamSize;
  const int lastStreamBytes = sizeof(DataType) * lastStreamSize;
  int offset;
  for (int i=0; i<nStreams; i++) {
    checkCuda( hipStreamCreate(&streams[i]) );
  }
  
  // Initialize the 1D grid and block dimensions
  uint blockSize = TPB;
  uint gridSize  = (streamSize+blockSize-1) / blockSize;

  // Copy and kernel execution
  t0 = time();
  for (int i=0; i<nStreams-1; i++) {
    offset= i*streamSize;
    checkCuda( hipMemcpyAsync(&deviceInput1[offset], &pinnedInput1[offset], streamBytes, hipMemcpyHostToDevice, streams[i]) );
    checkCuda( hipMemcpyAsync(&deviceInput2[offset], &pinnedInput2[offset], streamBytes, hipMemcpyHostToDevice, streams[i]) );
    vecAdd<<<gridSize, blockSize, 0, streams[i]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);
    checkCuda( hipMemcpyAsync(&pinnedOutput[offset], &deviceOutput[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]) );
  }
  int i = nStreams-1;
  offset= i*streamSize;
  gridSize = (lastStreamSize+blockSize-1) / blockSize;
  checkCuda( hipMemcpyAsync(&deviceInput1[offset], &pinnedInput1[offset], lastStreamBytes, hipMemcpyHostToDevice, streams[i]) );
  checkCuda( hipMemcpyAsync(&deviceInput2[offset], &pinnedInput2[offset], lastStreamBytes, hipMemcpyHostToDevice, streams[i]) );
  vecAdd<<<gridSize, blockSize, 0, streams[nStreams-1]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], lastStreamSize);
  checkCuda( hipMemcpyAsync(&pinnedOutput[offset], &deviceOutput[offset], lastStreamBytes, hipMemcpyDeviceToHost, streams[i]) );;
   
  for (int i=0; i<nStreams; i++) {
    checkCuda( hipStreamSynchronize(streams[i]) );
  }
  printf("Streams Synchronized");
  // Timing
  double vectorAddTiming = time() - t0;
  

  // Compare the output with the reference
  DataType max_diff = 1e-7;
  for (uint i=0; i<inputLength; i++) {
    if (abs(pinnedOutput[i]-resultRef[i])>1e-7) {
      printf("Index: %d, Error results differ more than maximum value (>%f)\n", i, max_diff);
      //printf("Host Calculated Value: %f\n", resultRef[i]);
      //printf("Device Calculated Value: %f\n", pinnedOutput[i]);
    }
  }
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  hipHostFree(pinnedInput1);
  hipHostFree(pinnedInput2);
  hipHostFree(pinnedOutput);


  // Measurement prints
  if (formattedPrint) {
    printf("%d, %11.8f, %11.8f\n", inputLength, cpuTiming, vectorAddTiming);
  }
  else {
    printf("Vector length: %d\n", inputLength);
    printf("CPU vector addition time: %f\n", cpuTiming);
    printf("GPU vector addition time: %f\t(incl. data transfer)\n", vectorAddTiming);
  }
  return 0;
}

